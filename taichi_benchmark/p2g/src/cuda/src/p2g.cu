#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <memory>

#include "eigen/Eigen/Dense"
#include "utils.h"

// Benchmark MPM2D
// dim, steps, dt = 2, 32, 1e-4

using Vector = Eigen::Vector2f;
using Matrix = Eigen::Matrix2f;
using Vectori = Eigen::Vector2i;
using Real = float;

// TODO global var
__device__ Real dt = 2e-4;
__device__ Real E = 400;
__device__ int dim = 2;
__device__ int steps = 32;
__device__ int neighbour = 9;
__device__ Real gravity = 9.8;
__device__ int bound = 3;
__device__ Real p_rho = 1.0;

Vector *x_dev;
Vector *v_dev;
Matrix *C_dev;
Real *J_dev;
Vector *grid_v_dev;
Real *grid_m_dev;

__global__ void init_kernel(Real *J) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  J[idx] = 1;
}

__global__ void reset_kernel(Vector *grid_v, Real *grid_m) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  grid_v[idx].setZero();
  grid_m[idx] = 0;
}

template <class R, class A> __device__ R narrow_cast(const A &a) {
  R r = R(a);
  if (A(r) != a)
    printf("warning: info loss in narrow_cast\n");
  return r;
}

__device__ Vectori get_offset(size_t idx) {
  Vectori offset;
  for (auto i = dim - 1; i >= 0; i--) {
    offset[i] = narrow_cast<int, size_t>(idx % 3);
    idx /= 3;
  }
  return offset;
}

__device__ Vectori get_indices(size_t idx, int n_grid) {
  Vectori indices;
  for (auto i = dim - 1; i >= 0; i--) {
    indices[i] = narrow_cast<int, size_t>(idx % n_grid);
    idx /= n_grid;
  }
  return indices;
}

__global__ void particle_to_grid_kernel(Vector *x, Vector *v, Matrix *C,
                                        Real *J, Vector *grid_v, Real *grid_m,
                                        Real dx, Real p_vol, Real p_mass,
                                        int n_grid) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  Vector Xp = x[idx] / dx;
  Vectori base = (Xp.array() - 0.5).cast<int>();
  Vector fx = Xp - base.cast<Real>();
  std::array<Vector, 3> w{0.5 * (1.5 - fx.array()).pow(2),
                          0.75 - (fx.array() - 1.0).pow(2),
                          0.5 * (fx.array() - 0.5).pow(2)};
  auto stress = -dt * 4 * E * p_vol * (J[idx] - 1) / std::pow(dx, 2);
  Matrix affine = Matrix::Identity() * stress + p_mass * C[idx];

  // Vector new_v = Vector::Zero();
  // Matrix new_C = Matrix::Zero();
  for (auto offset_idx = 0; offset_idx < neighbour; offset_idx++) {
    Vectori offset = get_offset(offset_idx);
    Vector dpos = (offset.cast<Real>() - fx) * dx;
    Real weight = 1.0;
    for (auto i = 0; i < dim; i++) {
      weight *= w[offset[i]][i];
    }
    // Vectori grid_idx_vector = base + offset;
    // auto grid_idx = 0;
    // for (auto i = 0; i < dim; i++) {
    //  grid_idx = grid_idx * n_grid + grid_idx_vector[i];
    //}
    // new_v += weight * grid_v[grid_idx];
    // new_C += 4.0 * weight * grid_v[grid_idx] * dpos.transpose() / pow(dx, 2);

    Vectori grid_idx_vector = base + offset;
    auto grid_idx = 0;
    for (auto i = 0; i < dim; i++) {
      grid_idx = grid_idx * n_grid + grid_idx_vector[i];
    }

    // update grid_v
    Vector grid_v_add = weight * (p_mass * v[idx] + affine * dpos);
    for (auto i = 0; i < dim; i++) {
      atomicAdd(&(grid_v[grid_idx][i]), grid_v_add[i]);
    }

    // update grid_m
    auto grid_m_add = weight * p_mass;
    atomicAdd(&(grid_m[grid_idx]), grid_m_add);
  }
  // v[idx] = new_v;
  // x[idx] += dt * v[idx];
  // J[idx] *= Real(1.0) + dt * new_C.trace();
  // C[idx] = new_C;
}

class MPM {
public:
  explicit MPM(int n_grid) : n_grid(n_grid) {
    dim = 2;
    steps = 32;
    n_particles = utils::power(n_grid, dim) / utils::power(2, dim - 1);
    neighbour = utils::power(3, dim);
    dx = 1.0 / n_grid;
    p_rho = 1.0;
    p_vol = utils::power(dx * 0.5, 2);
    p_mass = p_vol * p_rho;
    gravity = 9.8;
    bound = 3;
    E = 400;
  }

  void init() {
    hipFree(x_dev);
    hipFree(v_dev);
    hipFree(C_dev);
    hipFree(J_dev);
    hipFree(grid_v_dev);
    hipFree(grid_m_dev);

    hipMalloc(&x_dev, n_particles * sizeof(Vector));
    hipMalloc(&v_dev, n_particles * sizeof(Vector));
    hipMalloc(&C_dev, n_particles * sizeof(Matrix));
    hipMalloc(&J_dev, n_particles * sizeof(Real));
    hipMalloc(&grid_v_dev, utils::power(n_grid, dim) * sizeof(Vector));
    hipMalloc(&grid_m_dev, utils::power(n_grid, dim) * sizeof(Real));
    utils::cuda_check_error();

    // initialize x on the host and copy to the device
    auto x_host = std::make_unique<Vector[]>(n_particles);
    for (auto i = 0; i < n_particles; i++) {
      for (auto j = 0; j < dim; j++) {
        x_host[i][j] = Real(utils::rand_real());
      }
      x_host[i] = (x_host[i] * 0.4).array() + 0.15;
    }
    hipMemcpy(x_dev, x_host.get(), n_particles * sizeof(Vector),
               hipMemcpyHostToDevice);

    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);
    int block_dim{64};
    threads_per_block = std::min(block_dim, prop.maxThreadsPerBlock);
    auto block_num = utils::get_block_num(n_particles, threads_per_block);
    init_kernel<<<block_num, threads_per_block>>>(J_dev);
    utils::cuda_check_error();
  }

  void reset() {
    auto particle_block_num =
        utils::get_block_num(n_particles, threads_per_block);
    auto grid_block_num =
        utils::get_block_num(utils::power(n_grid, dim), threads_per_block);
    reset_kernel<<<grid_block_num, threads_per_block>>>(grid_v_dev, grid_m_dev);
  }

  void advance() {
    auto T = steps;
    auto particle_block_num =
        utils::get_block_num(n_particles, threads_per_block);
    auto grid_block_num =
        utils::get_block_num(utils::power(n_grid, dim), threads_per_block);
    while (T--) {
      particle_to_grid_kernel<<<particle_block_num, threads_per_block>>>(
          x_dev, v_dev, C_dev, J_dev, grid_v_dev, grid_m_dev, dx, p_vol, p_mass,
          n_grid);
    }
  }

  std::unique_ptr<Vector[]> to_numpy() {
    auto x_host = std::make_unique<Vector[]>(n_particles);
    hipMemcpy(x_host.get(), x_dev, n_particles * sizeof(Vector),
               hipMemcpyDeviceToHost);

    return x_host;
  }

  int get_n_particles() const { return n_particles; }

public:
  int dim = 2;
  int n_grid = 128;
  int steps = 32;
  int n_particles = utils::power(n_grid, dim) / utils::power(2, dim - 1);
  int neighbour = 9; // 2D
  Real dx = 1.0 / n_grid;
  Real p_rho = 1.0;
  Real p_vol = utils::power(dx * 0.5, 2);
  Real p_mass = p_vol * p_rho;
  Real gravity = 9.8;
  int bound = 3;
  Real E = 400;
  int threads_per_block;
};

int main(const int argc, const char **argv) {
  int n_grid = 128;
  if (argc > 1) {
    n_grid = atoi(argv[1]);
  }

  MPM *mpm = new MPM(n_grid);
  // skip first run
  mpm->init();
  mpm->reset();
  mpm->advance();
  auto x = mpm->to_numpy();

  int num_frames{1024};
  auto start_time = std::chrono::high_resolution_clock::now();
  for (auto runs = 0; runs < num_frames; runs++) {
    mpm->advance();
    // comment out to exclude D2H transfer
    // auto x = mpm->to_numpy();
  }
  hipDeviceSynchronize();
  auto end_time = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = end_time - start_time;

  float time_ms = diff.count() * 1000 / num_frames;
  printf("{\"n_particles\":%d, \"time_ms\": %f}\n", mpm->get_n_particles(),
         /*time_ms*/ time_ms);

  return 0;
}
