#include "reduce_sum.cuh"

int main(int argc, char **argv) {
    int num_items = 4096;
    if(argc > 1) num_items = std::atoi(argv[1]);
    float *h_in = new float [num_items];
    float *h_reference = new float;
    thrust::device_vector<float> u(num_items);
    Initialize(h_in, num_items);
    Solve(h_in, h_reference, num_items);

    cuErrCheck(hipMemcpy(thrust::raw_pointer_cast(&u[0]), h_in, sizeof(float) * num_items, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    float h_out = thrust::reduce(u.begin(), u.end(), 0.0f, thrust::plus<float>());
    hipDeviceSynchronize();
    cuErrCheck(hipGetLastError());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f", milliseconds);
    TestResult(&h_out, h_reference);

    delete[] h_in;
    delete[] h_reference;
}
