#include "reduce_sum.cuh"

using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);

int main(int argc, char **argv) {
    int num_items = 4096;
    if(argc > 1) num_items = std::atoi(argv[1]);
    float *d_in = nullptr;
    float *d_out = nullptr;
    float *h_in = new float [num_items];
    float *h_reference = new float;
    float *h_out = new float;
    Initialize(h_in, num_items);
    Solve(h_in, h_reference, num_items);

    hipMalloc(&d_in, num_items * sizeof(float));
    hipMalloc(&d_out, sizeof(float));
    cuErrCheck(hipMemcpy(d_in, h_in, sizeof(float) * num_items, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));
    hipDeviceSynchronize();
    cuErrCheck(hipGetLastError());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f", milliseconds);

    cuErrCheck(hipMemcpy(h_out, d_out, sizeof(float), hipMemcpyDeviceToHost));
    TestResult(h_out, h_reference);

    hipFree(d_in);
    hipFree(d_out);
    delete[] h_in;
    delete h_reference;
    delete h_out;
    return 0;
}