#include "hip/hip_runtime.h"
// Parallel Prefix Sum (Scan)
// Ref[0]:
// https://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/scan/doc/scan.pdf
// Ref[1]:
// https://github.com/NVIDIA/cuda-samples/blob/master/Samples/2_Concepts_and_Techniques/shfl_scan/shfl_scan.cu

// Last update: July 12, 2022
// Author: Bo Qiao

#include <cmath>
#include <cstdlib>
#include <iostream>
#include <limits>
#include <vector>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#define TYPE float

// Scan using shuffle instructions
__global__ void shfl_scan(TYPE *data, int len, TYPE *partial_sums = NULL) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx < len) {
    extern __shared__ TYPE sums[];
    const int warp_sz = 32;
    int lane_id = idx % warp_sz;
    int warp_id = threadIdx.x / warp_sz;

    TYPE value = data[idx];

    // intra-warp scan
    for (int i = 1; i <= 32; i *= 2) {
      TYPE n = __shfl_up_sync(0xffffffff, value, i);
      if (lane_id >= i) {
        value += n;
      }
    }

    // put warp scan results to smem
    if (threadIdx.x % warp_sz == warp_sz - 1) {
      sums[warp_id] = value;
    }
    __syncthreads();

    // inter-warp scan, use the first thread in the first warp
    if (warp_id == 0 && lane_id == 0) {
      for (int i = 1; i < blockDim.x / warp_sz; i++) {
        sums[i] += sums[i - 1];
      }
    }
    __syncthreads();

    // update data with warp_sums
    TYPE warp_sum = 0;
    if (warp_id > 0) {
      warp_sum = sums[warp_id - 1];
    }
    value += warp_sum;
    data[idx] = value;

    // update partial sums if applicable
    if (partial_sums != NULL && threadIdx.x == blockDim.x - 1) {
      partial_sums[blockIdx.x] = value;
    }
  }
}

// Aux function uniform add
__global__ void uniform_add(TYPE *data, TYPE *partial_sums, int len) {
  __shared__ TYPE buf;
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx <= len) {
    if (threadIdx.x == 0) {
      buf = partial_sums[blockIdx.x];
    }
    __syncthreads();
    data[idx] += buf;
  }
}

bool AreSameFloats(float a, float b) {
  return fabs(a - b) < std::numeric_limits<float>::epsilon();
}

// CPU golden results
// work-efficient sequential scan: exactly n additions, O(n)
bool compare_scan_golden(TYPE *output, TYPE *input, int len) {
  bool pass = true;
  TYPE sum = 0;
  for (int j = 0; j < len; j++) {
    sum += input[j];
    if (!AreSameFloats(sum, output[j])) {
      pass = false;
      std::cout << "[Fail] At pos " << j << ", golden " << sum << ", output "
                << output[j] << std::endl;
      break;
    }
  }
  return pass;
}

int main(int argc, char **argv) {
  // Parse input size
  int n_elements = 100000;
  if (argc > 1) {
    n_elements = atoi(argv[1]);
  }
  std::cout << "[Info] Number of elements: " << n_elements << std::endl;

  const int blockSize = 256;
  std::cout << "[Info] Block Size: " << blockSize << std::endl;
  int shmem_sz = blockSize / 32 * sizeof(TYPE);

  // Buffer allocations
  TYPE *h_data, *h_result, *h_result_golden;
  hipHostMalloc(reinterpret_cast<void **>(&h_data), sizeof(TYPE) * n_elements);
  hipHostMalloc(reinterpret_cast<void **>(&h_result),
                 sizeof(TYPE) * n_elements);
  hipHostMalloc(reinterpret_cast<void **>(&h_result_golden),
                 sizeof(TYPE) * n_elements);

  // Initialize host data
  for (size_t i = 0; i < n_elements; i++) {
    h_data[i] = 1.0;
    h_result_golden[i] = 0.0;
  }

  // Allocate device buffers to hold input and all intermediate partial sums
  std::vector<TYPE *> device_data;
  std::vector<int> device_data_ele_sz;
  for (int ne = n_elements; ne > 1; ne = (ne + blockSize - 1) / blockSize) {
    TYPE *data;
    hipMalloc(reinterpret_cast<void **>(&data), sizeof(TYPE) * ne);
    hipMemset(data, 0, sizeof(TYPE) * ne);
    device_data.push_back(data);
    device_data_ele_sz.push_back(ne);
  }

  // Copy data to device
  hipMemcpy(device_data[0], h_data, sizeof(TYPE) * n_elements,
             hipMemcpyHostToDevice);

  // Use events to time device execution
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Kernel launch
  hipEventRecord(start, 0);
  int it_id = 0;
  for (int ne = n_elements; ne > 1; ne = (ne + blockSize - 1) / blockSize) {
    int grid_size = (ne + blockSize - 1) / blockSize;
    if (grid_size == 1) {
      shfl_scan<<<1, blockSize, shmem_sz>>>(device_data.back(), ne);
    } else {
      shfl_scan<<<grid_size, blockSize, shmem_sz>>>(device_data[it_id], ne,
                                                    device_data[it_id + 1]);
    }
    it_id++;
  }

  // Add partial sums back
  for (int i = device_data_ele_sz.size() - 2; i >= 0; i--) {
    int p_grid_sz = (device_data_ele_sz[i] + blockSize - 1) / blockSize - 1;
    uniform_add<<<p_grid_sz, blockSize>>>(
        device_data[i] + blockSize, device_data[i + 1], device_data_ele_sz[i]);
  }

  hipEventRecord(stop, 0);

  // Copy result back to host
  hipMemcpy(h_result, device_data[0], sizeof(TYPE) * n_elements,
             hipMemcpyDeviceToHost);

  // Compare ground truth
  compare_scan_golden(h_result, h_data, n_elements);

  float time_ms = 0;
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_ms, start, stop);
  std::cout << "[Benchmark] Execution time: " << time_ms << " ms" << std::endl;
  std::cout << "[Benchmark] Achieved bandwidth: "
            << (sizeof(TYPE) * (float)n_elements / 1000000.0) / time_ms
            << " GB/s" << std::endl;

  // Thrust in-place scan as comparison
  thrust::device_vector<TYPE> d_values(h_data, h_data + n_elements);

  hipEventRecord(start, 0);
  thrust::inclusive_scan(d_values.begin(), d_values.end(), d_values.begin());
  hipEventRecord(stop, 0);

  thrust::host_vector<TYPE> h_thrust_result(d_values.begin(), d_values.end());
  compare_scan_golden(&h_thrust_result[0], h_data, n_elements);

  hipEventSynchronize(stop);
  time_ms = 0;
  hipEventElapsedTime(&time_ms, start, stop);
  std::cout << "[Benchmark] Thrust Execution time: " << time_ms << " ms"
            << std::endl;
  std::cout << "[Benchmark] Thrust Achieved bandwidth: "
            << (sizeof(TYPE) * (float)n_elements / 1000000.0) / time_ms
            << " GB/s" << std::endl;

  // Clean ups
  hipHostFree(h_data);
  hipHostFree(h_result);
  hipHostFree(h_result_golden);
  for (auto d_data : device_data) {
    hipFree(d_data);
  }
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return 0;
}
